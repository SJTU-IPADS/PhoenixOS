/*
 * Copyright 2024 The PhoenixOS Authors. All rights reserved.
 * 
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */


#include <hip/hip_runtime.h>
#include <iostream>

#include <math.h>
#include <stdio.h>
#include <stdlib.h>

#define CUDA_SAFECALL(call)                                                 \
    {                                                                       \
        call;                                                               \
        hipError_t err = hipGetLastError();                                 \
        if (hipSuccess != err) {                                           \
            fprintf(                                                        \
                stderr,                                                     \
                "Cuda error in function '%s' file '%s' in line %i : %s.\n", \
                #call, __FILE__, __LINE__, hipGetErrorString(err));        \
            fflush(stderr);                                                 \
            exit(EXIT_FAILURE);                                             \
        }                                                                   \
    }


// CUDA kernel. Each thread takes care of one element of c
__global__ void vecSec(double *a, double *b, double *c, int n) {
    // Get our global thread ID
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Make sure we do not go out of bounds
    if (id < n) c[id] = a[id] + b[id];
    if (id > n) c[id] = b[id] + a[id-1];
}

int main(int argc, char *argv[]) {
    // Size of vectors
    int n = 100000;
    if (argc > 1) n = atoi(argv[1]);

    // Host input vectors
    double *h_a;
    double *h_b;
    // Host output vector
    double *h_c;

    // Device input vectors
    double *d_a;
    double *d_b;
    // Device output vector
    double *d_c;

    // Size, in bytes, of each vector
    size_t bytes = n * sizeof(double);

    // Allocate memory for each vector on host
    h_a = (double *)malloc(bytes);
    h_b = (double *)malloc(bytes);
    h_c = (double *)malloc(bytes);

    // Allocate memory for each vector on GPU
    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);

    int i;
    // Initialize vectors on host
    for (i = 0; i < n; i++) {
        h_a[i] = sin(i) * sin(i);
        h_b[i] = cos(i) * cos(i);
        h_c[i] = 0;
    }

    // Copy host vectors to device
    hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_c, h_c, bytes, hipMemcpyHostToDevice);

    int blockSize, gridSize;

    // Number of threads in each thread block
    blockSize = 1024;

    // Number of thread blocks in grid
    gridSize = (int)ceil((float)n / blockSize);

    // Execute the kernel
    CUDA_SAFECALL((vecSec<<<gridSize, blockSize>>>(d_a, d_b, d_c, n)));
    hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);

    // Execute the kernel
    CUDA_SAFECALL((vecSec<<<gridSize, blockSize>>>(d_a, d_b, d_c, n)));
    hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);

    // Execute the kernel
    CUDA_SAFECALL((vecSec<<<gridSize, blockSize>>>(d_a, d_b, d_c, n)));
    hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);

    // Execute the kernel
    CUDA_SAFECALL((vecSec<<<gridSize, blockSize>>>(d_a, d_b, d_c, n)));
    hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);

    // Execute the kernel
    CUDA_SAFECALL((vecSec<<<gridSize, blockSize>>>(d_a, d_b, d_c, n)));
    hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);

    // Sum up vector c and print result divided by n, this should equal 1 within
    // error
    double sum = 0;
    for (i = 0; i < n; i++) sum += h_c[i];
    printf("Final sum = %f; sum/n = %f (should be ~1)\n", sum, sum / n);

    // Release device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    // Release host memory
    free(h_a);
    free(h_b);
    free(h_c);

    return 0;
}
