#include <iostream>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#define ROUND_UP(size, aligned_size)    ((size + aligned_size - 1) / aligned_size) * aligned_size;

static inline void
checkRtError(hipError_t res, const char *tok, const char *file, unsigned line)
{
    if (res != hipSuccess) {
        std::cerr << file << ':' << line << ' ' << tok
                  << "failed (" << (unsigned)res << "): " << hipGetErrorString(res) << std::endl;
        abort();
    }
}

#define CHECK_RT(x) checkRtError(x, #x, __FILE__, __LINE__);

static inline void
checkDrvError(hipError_t res, const char *tok, const char *file, unsigned line)
{
    if (res != hipSuccess) {
        const char *errStr = NULL;
        (void)hipDrvGetErrorString(res, &errStr);
        std::cerr << file << ':' << line << ' ' << tok
                  << "failed (" << (unsigned)res << "): " << errStr << std::endl;
        abort();
    }
}

#define CHECK_DRV(x) checkDrvError(x, #x, __FILE__, __LINE__);

int main(){
    hipMemAllocationProp prop = {};
    size_t sz, aligned_sz;
    hipMemGenericAllocationHandle_t hdl;
    hipMemAccessDesc accessDesc;
    hipDeviceptr_t ptr, req_ptr;
    int dev = 0;
    uint64_t size = 8192;

    // init runtime
    CHECK_RT(hipSetDevice(dev));

    prop.type = hipMemAllocationTypePinned;
    prop.location.type = hipMemLocationTypeDevice;
    prop.location.id = dev;
    accessDesc.location = prop.location;
    accessDesc.flags = hipMemAccessFlagsProtReadWrite;

    CHECK_DRV(hipMemGetAllocationGranularity(&aligned_sz, &prop, hipMemAllocationGranularityMinimum));
    sz = ROUND_UP(size, aligned_sz);

    // give a hint (48-bit GPU memory address)
    req_ptr = 0x555500000000;
    CHECK_DRV(hipMemAddressReserve(&ptr, sz, 0ULL, req_ptr, 0ULL));
    CHECK_DRV(hipMemCreate(&hdl, sz, &prop, 0));
    CHECK_DRV(hipMemMap(ptr, sz, 0ULL, hdl, 0ULL));
    CHECK_DRV(hipMemSetAccess(ptr, sz, &accessDesc, 1ULL));

    printf("cuda vm: %p\n", ptr);

    CHECK_DRV(hipMemUnmap(ptr, sz));
    CHECK_DRV(hipMemAddressFree(ptr, sz));
    CHECK_DRV(hipMemRelease(hdl));
}
