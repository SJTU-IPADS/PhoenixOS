/*
 * Copyright 2024 The PhoenixOS Authors. All rights reserved.
 * 
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <iostream>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#define ROUND_UP(size, aligned_size)    ((size + aligned_size - 1) / aligned_size) * aligned_size;

static inline void
checkRtError(hipError_t res, const char *tok, const char *file, unsigned line)
{
    if (res != hipSuccess) {
        std::cerr << file << ':' << line << ' ' << tok
                  << "failed (" << (unsigned)res << "): " << hipGetErrorString(res) << std::endl;
        abort();
    }
}

#define CHECK_RT(x) checkRtError(x, #x, __FILE__, __LINE__);

static inline void
checkDrvError(hipError_t res, const char *tok, const char *file, unsigned line)
{
    if (res != hipSuccess) {
        const char *errStr = NULL;
        (void)hipDrvGetErrorString(res, &errStr);
        std::cerr << file << ':' << line << ' ' << tok
                  << "failed (" << (unsigned)res << "): " << errStr << std::endl;
        abort();
    }
}

#define CHECK_DRV(x) checkDrvError(x, #x, __FILE__, __LINE__);

int main(){
    hipMemAllocationProp prop = {};
    size_t sz, aligned_sz;
    hipMemGenericAllocationHandle_t hdl;
    hipMemAccessDesc accessDesc;
    hipDeviceptr_t ptr, req_ptr;
    int dev = 0;
    uint64_t size = 64 * 2097152;

    // init runtime
    CHECK_RT(hipSetDevice(dev));

    prop.type = hipMemAllocationTypePinned;
    prop.location.type = hipMemLocationTypeDevice;
    prop.location.id = dev;
    accessDesc.location = prop.location;
    accessDesc.flags = hipMemAccessFlagsProtReadWrite;

    CHECK_DRV(hipMemGetAllocationGranularity(&aligned_sz, &prop, hipMemAllocationGranularityMinimum));
    sz = ROUND_UP(size, aligned_sz);

    // give a hint (48-bit GPU memory address)
    req_ptr = 0x555000000000;
    CHECK_DRV(hipMemAddressReserve(&ptr, sz, 4*1024*1024, req_ptr, 0ULL));
        
    printf("cuda vm: %p, aligned_sz: %lu\n", ptr, aligned_sz);

    CHECK_DRV(hipMemCreate(&hdl, sz, &prop, 0));
    CHECK_DRV(hipMemMap(ptr+32*2097152, sz, 0ULL, hdl, 0ULL));
    CHECK_DRV(hipMemSetAccess(ptr+32*2097152, sz, &accessDesc, 1ULL));
    CHECK_DRV(hipMemUnmap(ptr+32*2097152, sz));
     CHECK_DRV(hipMemAddressFree(ptr+32*2097152, sz));
    CHECK_DRV(hipMemRelease(hdl));
}
