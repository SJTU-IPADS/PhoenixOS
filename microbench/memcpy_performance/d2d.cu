#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <fstream>

#include <sys/resource.h>
#include <stdint.h>
#include <hip/hip_runtime_api.h>
#include <nvToolsExt.h>

#include "mb_common/ticks.h"

#define PROFILING_FILE_PATH "/root/microbench/memcpy_performance/build/profile.txt"


int main(){
    // initialize constants
    std::vector<uint64_t> buffer_sizes;
    constexpr uint64_t nb_buffers = 28;
    uint64_t i, s_tick, e_tick, max_buffer_size;
    double duration_us;
    void *ptr;
    std::vector<void*> mems;
    void *dev_dest_buf;
    std::ofstream output_file;
    hipError_t cuda_rt_retval;

    struct rusage s_r_usage, e_r_usage;

    output_file.open(PROFILING_FILE_PATH, std::fstream::in | std::fstream::out | std::fstream::trunc);
    
    // set buffer sizes
    for(i=0; i<nb_buffers; i++){
        buffer_sizes.push_back(1<<i);
    }
    max_buffer_size = buffer_sizes[nb_buffers-1];

    // set dst device buffer
    cuda_rt_retval = hipMalloc(&dev_dest_buf, max_buffer_size);
    if(cuda_rt_retval != hipSuccess){
        printf("failed hipMalloc: %d\n", cuda_rt_retval);
        exit(1);
    }

    // malloc corresponding src buffer
    for(i=0; i<nb_buffers; i++){
        if(hipSuccess != hipMalloc(&ptr, buffer_sizes[i])){
            printf("failed malloc %lu\n", i);
            exit(1);
        }
        mems.push_back(ptr);
    }

    // warmup
    hipMemcpyAsync(dev_dest_buf, mems[1], buffer_sizes[1], hipMemcpyDeviceToDevice, 0);
    hipStreamSynchronize(0);

    // measure
    for(i=0; i<nb_buffers; i++){
        if(getrusage(RUSAGE_SELF, &s_r_usage) != 0){
            printf("failed getrusage at %lu\n", i);
            exit(1);
        }

        s_tick = get_tsc();
        if(hipSuccess != hipMemcpy(
            dev_dest_buf,
            mems[i],
            buffer_sizes[i],
            hipMemcpyDeviceToDevice
        )){
            printf("failed hipMemcpyAsync at %lu\n", i);
            exit(1);
        }
        e_tick = get_tsc();

        if(getrusage(RUSAGE_SELF, &e_r_usage) != 0){
            printf("failed getrusage at %lu\n", i);
            exit(1);
        }

        duration_us = POS_TSC_RANGE_TO_USEC(e_tick, s_tick);
        printf(
            "copy duration: %lf us, size: %lu Bytes, bw: %lf Mbps, page fault: %ld (major), %ld (minor)\n",
            duration_us, buffer_sizes[i], (double)(buffer_sizes[i]) / duration_us,
            e_r_usage.ru_majflt - s_r_usage.ru_majflt,
            e_r_usage.ru_minflt - s_r_usage.ru_minflt
        );
        output_file << duration_us << "," << buffer_sizes[i] << std::endl;
    }

    for(i=0; i<nb_buffers; i++){
        hipFree(mems[i]);
    }
    hipHostFree(dev_dest_buf);

    output_file.close();

    return 0;
}
