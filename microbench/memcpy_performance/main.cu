#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <fstream>

#include <sys/resource.h>
#include <stdint.h>
#include <hip/hip_runtime_api.h>
#include <nvToolsExt.h>

#include "mb_common/ticks.h"

#define PROFILING_FILE_PATH "/root/memcpy_performance/profile.txt"


int main(){
    // initialize constants
    std::vector<uint64_t> buffer_sizes;
    constexpr uint64_t nb_buffers = 28;
    uint64_t i, s_tick, e_tick, max_buffer_size;
    double duration_us;
    void *ptr;
    std::vector<void*> mems;
    std::vector<float> host_data;
    std::ofstream output_file;

    struct rusage s_r_usage, e_r_usage;

    output_file.open(PROFILING_FILE_PATH, std::fstream::in | std::fstream::out | std::fstream::trunc);
    
    // set buffer sizes
    for(i=0; i<nb_buffers; i++){
        buffer_sizes.push_back(1<<i);
    }
    max_buffer_size = buffer_sizes[nb_buffers-1];

    // set host buffer
    host_data.reserve(max_buffer_size);
    for(i=0; i<max_buffer_size; i++){
        host_data.push_back(1.0f);
    }

    // malloc corresponding buffer
    for(i=0; i<nb_buffers; i++){
        if(hipSuccess != hipMalloc(&ptr, buffer_sizes[i])){
            printf("failed malloc %lu\n", i);
            exit(1);
        }
        mems.push_back(ptr);
    }

    // warmup
    hipMemcpyAsync(host_data.data(), mems[1], buffer_sizes[1], hipMemcpyDeviceToHost, 0);
    hipStreamSynchronize(0);

    // measure
    for(i=0; i<nb_buffers; i++){
        if(getrusage(RUSAGE_SELF, &s_r_usage) != 0){
            printf("failed getrusage at %lu\n", i);
            exit(1);
        }

        s_tick = get_tsc();
        if(hipSuccess != hipMemcpyAsync(
            host_data.data(),
            mems[i],
            buffer_sizes[i],
            hipMemcpyDeviceToHost,
            0
        )){
            printf("failed hipMemcpyAsync at %lu\n", i);
            exit(1);
        }
        e_tick = get_tsc();

        if(getrusage(RUSAGE_SELF, &e_r_usage) != 0){
            printf("failed getrusage at %lu\n", i);
            exit(1);
        }

        duration_us = POS_TSC_RANGE_TO_USEC(e_tick, s_tick);
        printf(
            "copy duration: %lf us, size: %lu Bytes, bw: %lf Mbps, page fault: %ld (major), %ld (minor)\n",
            duration_us, buffer_sizes[i], (double)(buffer_sizes[i]) / duration_us,
            e_r_usage.ru_majflt - s_r_usage.ru_majflt,
            e_r_usage.ru_minflt - s_r_usage.ru_minflt
        );
        output_file << duration_us << "," << buffer_sizes[i] << std::endl;
    }

    for(i=0; i<nb_buffers; i++){
        hipFree(mems[i]);
    }

    output_file.close();

    return 0;
}
