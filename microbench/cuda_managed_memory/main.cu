#include <iostream>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>


#define ROUND_UP(size, aligned_size)    ((size + aligned_size - 1) / aligned_size) * aligned_size;


static inline void
checkRtError(hipError_t res, const char *tok, const char *file, unsigned line)
{
    if (res != hipSuccess) {
        std::cerr << file << ':' << line << ' ' << tok
                  << "failed (" << (unsigned)res << "): " << hipGetErrorString(res) << std::endl;
        abort();
    }
}


#define CHECK_RT(x) checkRtError(x, #x, __FILE__, __LINE__);


__global__ void test_kernel(int *output_mem, size_t mem_size){
    int i;
    int fatten_id = blockIdx.x * blockDim.x + threadIdx.x;
    
    for(i=fatten_id; i<mem_size; i+=blockDim.x){
        if(fatten_id < mem_size)
            output_mem[i] = 2;
    }
}


int main(){
    int *read_vector;

    uint64_t vector_size = 8192;

    // allocate vector for reading test
    CHECK_RT(hipMallocManaged((void**)&read_vector, vector_size));

    // launch kernel
    test_kernel<<<1, 256>>>(read_vector, vector_size);

    // free
    CHECK_RT(hipFree(read_vector));
}
