#include "hip/hip_runtime.h"
#include"test_cuda/test_cuda_common.h"

__global__ void dummyKernel(int *data) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    data[idx] = idx;
}

TEST_F(PhOSCudaTest, hipFuncGetAttributes) {
    cudaFunctionAttributes attr;
    void *func_ptr = (void*)dummyKernel;

    hipError_t cuda_retval = (hipError_t)this->_ws->pos_process(
        /* api_id */ PosApiIndex_cudaFuncGetAttributes,
        /* uuid */ this->_clnt->id,
        /* param_desps */ {
            { .value = &attr, .size = sizeof(cudaFunctionAttributes) },
            { .value = &func_ptr, .size = sizeof(void*) }
        }
    );

    EXPECT_EQ(hipSuccess, cuda_retval);

    std::cout << "sharedSizeBytes: " << attr.sharedSizeBytes << std::endl;
    std::cout << "numRegs: " << attr.numRegs << std::endl;
    std::cout << "maxThreadsPerBlock: " << attr.maxThreadsPerBlock << std::endl;


    EXPECT_GT(attr.maxThreadsPerBlock, 0);
    EXPECT_GE(attr.numRegs, 0);
}
