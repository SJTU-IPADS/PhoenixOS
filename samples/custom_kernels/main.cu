#include "hip/hip_runtime.h"
#include <iostream>

#include <assert.h>
#include <unistd.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include "pos/include/common.h"
#include "pos/include/log.h"
#include "pos/include/utils/timestamp.h"

static inline void
checkRtError(hipError_t res, const char *tok, const char *file, unsigned line)
{
    if (res != hipSuccess) {
        std::cerr << file << ':' << line << ' ' << tok
                  << "failed (" << (unsigned)res << "): " << hipGetErrorString(res) << std::endl;
        abort();
    }
}
#define CHECK_RT(x) checkRtError(x, #x, __FILE__, __LINE__);

#define MB(x)   ((size_t) (x) << 20)

// mock kernel execution duration
template<unsigned us>
__device__ void __sleep_us() {    
    __nanosleep(us*1000);
}

__global__ void kernel_1(const float* in_a, float* out_a, float* out_b, float* out_c, int len){ 
    __sleep_us<5>();
}
__global__ void kernel_2(const float* in_a, const float* in_b, const float* in_c, float* out_a, int len){
   __sleep_us<10>();
}
__global__ void kernel_3(const float* in_a, float* out_a, int len){ 
    __sleep_us<15>();
}
__global__ void kernel_4(const float* in_a, const float* in_b, float* out_a, int len){ 
    __sleep_us<15>();
}

int main(){
    uint64_t i, j;
    float *mem_1, *mem_2, *mem_3, *mem_4, *mem_5, *mem_6;

    uint64_t s_tick, e_tick;

    // hipDeviceProp_t prop;

    // CHECK_RT(hipGetDeviceProperties(&prop, 0));
    // printf("device clock rate: %d\n", prop.clockRate);
    // assert(prop.clockRate == CLOCK_RATE);

    CHECK_RT(hipMalloc(&mem_1, MB(16)));
    CHECK_RT(hipMalloc(&mem_2, MB(64)));
    CHECK_RT(hipMalloc(&mem_3, MB(32)));
    CHECK_RT(hipMalloc(&mem_4, MB(128)));
    CHECK_RT(hipMalloc(&mem_5, MB(8)));
    CHECK_RT(hipMalloc(&mem_6, MB(2)));

    s_tick = POSUtilTimestamp::get_tsc();

    for(i=0; i<8; i++){
        for(j=0; j<512; j++){
            kernel_1<<<1,128>>>(mem_1, mem_2, mem_3, mem_4, 0);
            kernel_2<<<1,128>>>(mem_2, mem_4, mem_1, mem_3, 0);
            kernel_3<<<1,128>>>(mem_4, mem_3, 0);
            kernel_4<<<1,128>>>(mem_3, mem_5, mem_6, 0);
            kernel_3<<<1,128>>>(mem_6, mem_4, 0);
            kernel_1<<<1,128>>>(mem_2, mem_4, mem_6, mem_1, 0);
        }
        CHECK_RT(hipStreamSynchronize(0));
        usleep(1000);
    }

    e_tick = POSUtilTimestamp::get_tsc();

    POS_LOG("E2E Latency: %lf us", POS_TSC_TO_USEC(e_tick-s_tick));

    CHECK_RT(hipFree(mem_1));
    CHECK_RT(hipFree(mem_2));
    CHECK_RT(hipFree(mem_3));
    CHECK_RT(hipFree(mem_4));
    CHECK_RT(hipFree(mem_5));
    CHECK_RT(hipFree(mem_6));

    return 0;
}
