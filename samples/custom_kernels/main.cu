#include "hip/hip_runtime.h"
#include <iostream>

#include <assert.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

static inline void
checkRtError(hipError_t res, const char *tok, const char *file, unsigned line)
{
    if (res != hipSuccess) {
        std::cerr << file << ':' << line << ' ' << tok
                  << "failed (" << (unsigned)res << "): " << hipGetErrorString(res) << std::endl;
        abort();
    }
}
#define CHECK_RT(x) checkRtError(x, #x, __FILE__, __LINE__);

#define MB(x)   ((size_t) (x) << 20)

// mock kernel execution duration
#define CLOCK_RATE 2250000  // 4060
__device__ void __sleep_us(float t) {    
    clock_t t0 = clock64();
    clock_t t1 = t0;
    while ((t1 - t0)/(double(CLOCK_RATE)/(1000.0f)) < t)
        t1 = clock64();
}


__global__ void kernel_1(const float* in_a, float* out_a, float* out_b, float* out_c, int len){ __sleep_us(50.0f); }
__global__ void kernel_2(const float* in_a, const float* in_b, const float* in_c, float* out_a, int len){ __sleep_us(200.0f); }


int main(){
    uint64_t i;
    float *mem_1, *mem_2, *mem_3, *mem_4, *mem_5, *mem_6;
    hipDeviceProp_t prop;

    CHECK_RT(hipGetDeviceProperties(&prop, 0));
    printf("device clock rate: %d\n", prop.clockRate);
    assert(prop.clockRate == CLOCK_RATE);

    CHECK_RT(hipMalloc(&mem_1, MB(128)));
    CHECK_RT(hipMalloc(&mem_2, MB(128)));
    CHECK_RT(hipMalloc(&mem_3, MB(128)));
    CHECK_RT(hipMalloc(&mem_4, MB(128)));
    CHECK_RT(hipMalloc(&mem_5, MB(128)));
    CHECK_RT(hipMalloc(&mem_6, MB(128)));

    for(i=0; i<64; i++){
        kernel_1<<<1,128>>>(mem_1, mem_2, mem_3, mem_4, 0);
        kernel_2<<<1,128>>>(mem_1, mem_2, mem_3, mem_4, 0);
    }
    CHECK_RT(hipStreamSynchronize(0));

    CHECK_RT(hipFree(mem_1));
    CHECK_RT(hipFree(mem_2));
    CHECK_RT(hipFree(mem_3));
    CHECK_RT(hipFree(mem_4));
    CHECK_RT(hipFree(mem_5));
    CHECK_RT(hipFree(mem_6));

    return 0;
}
