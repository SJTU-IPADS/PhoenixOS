#include "hip/hip_runtime.h"
#include <iostream>

#include <assert.h>
#include <unistd.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include "pos/include/common.h"
#include "pos/include/log.h"
#include "pos/include/utils/timestamp.h"

static inline void
checkRtError(hipError_t res, const char *tok, const char *file, unsigned line)
{
    if (res != hipSuccess) {
        std::cerr << file << ':' << line << ' ' << tok
                  << "failed (" << (unsigned)res << "): " << hipGetErrorString(res) << std::endl;
        abort();
    }
}
#define CHECK_RT(x) checkRtError(x, #x, __FILE__, __LINE__);

#define MB(x)   ((size_t) (x) << 20)

typedef struct mock_dev_ptr_struct {
    int nothing_1;
    int nothing_2;
    bool nothing_3;
    void *dev_ptr_1;
    void *dev_ptr_2;
    bool nothing_4;
    int nothing_5;
} mock_dev_ptr_struct_t;

__global__ void kernel_with_struct_input(const float* in_a, float* out_a, mock_dev_ptr_struct_t mock_struct_input, int len){ 
    ((float*)(mock_struct_input.dev_ptr_1))[0] = in_a[0];
}

int main(){
    float *mem_1, *mem_2, *mem_3, *mem_4, *mem_5, *mem_6;
    mock_dev_ptr_struct_t struct_input;
    uint64_t s_tick, e_tick;

    // hipDeviceProp_t prop;

    // CHECK_RT(hipGetDeviceProperties(&prop, 0));
    // printf("device clock rate: %d\n", prop.clockRate);
    // assert(prop.clockRate == CLOCK_RATE);

    CHECK_RT(hipMalloc(&mem_1, MB(16)));
    CHECK_RT(hipMalloc(&mem_2, MB(64)));
    CHECK_RT(hipMalloc(&mem_3, MB(32)));
    CHECK_RT(hipMalloc(&mem_4, MB(128)));
    CHECK_RT(hipMalloc(&mem_5, MB(8)));
    CHECK_RT(hipMalloc(&mem_6, MB(2)));

    struct_input.dev_ptr_1 = mem_1;
    struct_input.dev_ptr_2 = mem_2;
    
    POS_LOG("mem_1: %p, mem_2: %p", mem_1, mem_2);

    POS_LOG("sizeof mock_dev_ptr_struct_t: %lu bytes", sizeof(mock_dev_ptr_struct));

    s_tick = POSUtilTimestamp::get_tsc();

    kernel_with_struct_input<<<1,128>>>(mem_1, mem_2, struct_input, 0);
    kernel_with_struct_input<<<1,128>>>(mem_1, mem_2, struct_input, 0);
    
    e_tick = POSUtilTimestamp::get_tsc();

    POS_LOG("E2E Latency: %lf us", POS_TSC_TO_USEC(e_tick-s_tick));

    CHECK_RT(hipFree(mem_1));
    CHECK_RT(hipFree(mem_2));
    CHECK_RT(hipFree(mem_3));
    CHECK_RT(hipFree(mem_4));
    CHECK_RT(hipFree(mem_5));
    CHECK_RT(hipFree(mem_6));

    return 0;
}
