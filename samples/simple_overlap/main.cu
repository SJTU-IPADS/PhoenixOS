#include "hip/hip_runtime.h"
#include <iostream>

#include <assert.h>
#include <unistd.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include "pos/include/common.h"
#include "pos/include/log.h"
#include "pos/include/utils/timestamp.h"

static inline void
checkRtError(hipError_t res, const char *tok, const char *file, unsigned line)
{
    if (res != hipSuccess) {
        std::cerr << file << ':' << line << ' ' << tok
                  << "failed (" << (unsigned)res << "): " << hipGetErrorString(res) << std::endl;
        abort();
    }
}
#define CHECK_RT(x) checkRtError(x, #x, __FILE__, __LINE__);

#define MB(x)   ((size_t) (x) << 20)
#define GB(x)   ((size_t) (x) << 30)

__global__ void kernel_1(const float* in_a, float* out_a){ 
    
#define DEVICE_FREQUENCY_1_1000 1380
#define SLEEP_MS_TO_CYCLES(x)   x * DEVICE_FREQUENCY_1_1000000

    out_a[1] = in_a[0];
    out_a[2] = in_a[1];
}


int main(){
    float *mem_1, *mem_2;

    CHECK_RT(hipMalloc(&mem_1, GB(2)));
    CHECK_RT(hipMalloc(&mem_2, GB(4)));

    for(uint64_t i=0; i<50; i++){
        for(uint64_t j=0; j<200; j++){
            kernel_1<<<1,128>>>(mem_1, mem_1);
        }
        POSUtilTimestamp::delay_us(500);

        for(uint64_t j=0; j<200; j++){
            kernel_1<<<1,128>>>(mem_2, mem_2);
        }
        POSUtilTimestamp::delay_us(500);
    }
}
